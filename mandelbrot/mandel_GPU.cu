#include "hip/hip_runtime.h"
/*
* mandel_GPU.cu
*
*  Created on: 17.12.2014
*      Author: raub
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "mandel_GPU.cuh"

dim3 ThreadsPerBlock(16,16);
dim3 BlocksPerGrid;

//o~--------------------------------------------------------------------~o//
__global__ void mandel_gpu(uchar4* img, CRange range, unsigned int MaxIter)
//o~--------------------------------------------------------------------~o//
{
    unsigned int x = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(x < range.window.x && y < range.window.y)
    {
        unsigned int i = x + y*blockDim.y*gridDim.x;

        float cx = range.xmin + range.xstep * (float)x;
        float cy = range.ymin + range.ystep * (float)y;

        float px = 0;
        float py = 0;
        float tmp = 0;

        unsigned int n = 0;
        while(n++ <= MaxIter && px*px + py*py < 4)
        {
            tmp = px*px - py*py + cx;
            py = 2 * px * py + cy;
            px = tmp;
        }

        img[i].w = 255;
        if(n >= MaxIter)
        {
            img[i].x = 255;
            img[i].y = 255;
            img[i].z = 255;
        }
        else
        {
            img[i].x = 0;
            img[i].y = 0;
            img[i].z = 0;
        }
    }
}


//o~--------------------------------------------------------------------~o//
__global__ void simpleframe(uchar4* img, CRange range)
//o~--------------------------------------------------------------------~o//
{
    unsigned int x = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y*blockDim.y;

    unsigned int i = x + y*blockDim.y*gridDim.x;

    img[i].x = 0;
    img[i].y = 0;
    img[i].z = 0;
    img[i].w = 255;

    if(x == 0 || x == range.window.x-1 || y == 0 || y == range.window.y-1)
    {
        img[i].x = 255;
    }
}


//o~--------------------------------------------------------------------~o//
void calcGrid(dim3 windowSize)
//o~--------------------------------------------------------------------~o//
{
    if(ThreadsPerBlock.x >= windowSize.x)
    {
        BlocksPerGrid.x = 1;
    }
    else
    {
        BlocksPerGrid.x = (windowSize.x + (ThreadsPerBlock.x-1)) / ThreadsPerBlock.x;
    }

    if(ThreadsPerBlock.y >= windowSize.y)
    {
        BlocksPerGrid.y = 1;
    }
    else
    {
        BlocksPerGrid.y = (windowSize.y + (ThreadsPerBlock.y-1)) / ThreadsPerBlock.y;
    }

    //  printf("Window: %d x %d => Grid: (%d x %d) x (%d x %d)\n", windowSize.x, windowSize.y,
    //                                                             BlocksPerGrid.x, BlocksPerGrid.y,
    //                                                             ThreadsPerBlock.x, ThreadsPerBlock.y);
}



//o~--------------------------------------------------------------------~o//
void render(uchar4* img, CRange range, unsigned int MaxIter)
//o~--------------------------------------------------------------------~o//
{
    mandel_gpu<<<BlocksPerGrid, ThreadsPerBlock>>>(img, range, MaxIter);
    //simpleframe<<<BlocksPerGrid, ThreadsPerBlock>>>(img, range);
}
